#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <array>
#include <complex>
#include <cstdio>
#include <numeric>
#include <vector>

#include <getopt.h>

#include <mpi.h>

#include <hip/hip_runtime.h>

#include "cudecomp.h"

// Error checking macros
#define CHECK_CUDECOMP_EXIT(call)                                                                                      \
  do {                                                                                                                 \
    cudecompResult_t err = call;                                                                                       \
    if (CUDECOMP_RESULT_SUCCESS != err) {                                                                              \
      fprintf(stderr, "%s:%d CUDECOMP error. (error code %d)\n", __FILE__, __LINE__, err);                             \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (false)

#define CHECK_CUDA_EXIT(call)                                                                                          \
  do {                                                                                                                 \
    hipError_t err = call;                                                                                            \
    if (hipSuccess != err) {                                                                                          \
      fprintf(stderr, "%s:%d CUDA error. (%s)\n", __FILE__, __LINE__, hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (false)

#define CHECK_MPI_EXIT(call)                                                                                           \
  {                                                                                                                    \
    int err = call;                                                                                                    \
    if (0 != err) {                                                                                                    \
      char error_str[MPI_MAX_ERROR_STRING];                                                                            \
      int len;                                                                                                         \
      MPI_Error_string(err, error_str, &len);                                                                          \
      if (error_str) {                                                                                                 \
        fprintf(stderr, "%s:%d MPI error. (%s)\n", __FILE__, __LINE__, error_str);                                     \
      } else {                                                                                                         \
        fprintf(stderr, "%s:%d MPI error. (error code %d)\n", __FILE__, __LINE__, err);                                \
      }                                                                                                                \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  }                                                                                                                    \
  while (false)

// CUDA kernel to demonstrate pencil data access on device.
__global__ void initialize_pencil(double* data, cudecompPencilInfo_t pinfo) {

  int64_t l = blockIdx.x * blockDim.x + threadIdx.x;

  if (l > pinfo.size) return;

  int i = l % pinfo.shape[0];
  int j = l / pinfo.shape[0] % pinfo.shape[1];
  int k = l / (pinfo.shape[0] * pinfo.shape[1]);

  int gx[3];
  gx[pinfo.order[0]] = i + pinfo.lo[0];
  gx[pinfo.order[1]] = j + pinfo.lo[1];
  gx[pinfo.order[2]] = k + pinfo.lo[2];

  gx[pinfo.order[0]] -= pinfo.halo_extents[pinfo.order[0]];
  gx[pinfo.order[1]] -= pinfo.halo_extents[pinfo.order[1]];
  gx[pinfo.order[2]] -= pinfo.halo_extents[pinfo.order[2]];

  data[i] = gx[0] + gx[1] + gx[2];
}

int main(int argc, char** argv) {

  // Initialize MPI and start up cuDecomp
  CHECK_MPI_EXIT(MPI_Init(nullptr, nullptr));
  int rank, nranks;
  CHECK_MPI_EXIT(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
  CHECK_MPI_EXIT(MPI_Comm_size(MPI_COMM_WORLD, &nranks));

  if (nranks != 4) {
    fprintf(stderr, "ERROR: This example requires 4 ranks to run. Exiting...\n");
    exit(EXIT_FAILURE);
  }

  MPI_Comm local_comm;
  MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &local_comm);
  int local_rank;
  MPI_Comm_rank(local_comm, &local_rank);
  CHECK_CUDA_EXIT(hipSetDevice(local_rank));

  cudecompHandle_t handle;
  CHECK_CUDECOMP_EXIT(cudecompInit(&handle, MPI_COMM_WORLD));

  // Create cuDecomp grid descriptor
  cudecompGridDescConfig_t config;
  CHECK_CUDECOMP_EXIT(cudecompGridDescConfigSetDefaults(&config));

  config.pdims[0] = 2; // P_rows
  config.pdims[1] = 2; // P_cols

  config.gdims[0] = 64; // X
  config.gdims[1] = 64; // Y
  config.gdims[2] = 64; // Z

  config.transpose_comm_backend = CUDECOMP_TRANSPOSE_COMM_MPI_P2P;
  config.halo_comm_backend = CUDECOMP_HALO_COMM_MPI;

  config.transpose_axis_contiguous[0] = true;
  config.transpose_axis_contiguous[1] = true;
  config.transpose_axis_contiguous[2] = true;

  cudecompGridDesc_t grid_desc;
  CHECK_CUDECOMP_EXIT(cudecompGridDescCreate(handle, &grid_desc, &config, nullptr));

  // Print information on configuration
  if (rank == 0) {
    printf("running on %d x %d process grid...\n", config.pdims[0], config.pdims[1]);
    printf("running using %s transpose backend...\n",
           cudecompTransposeCommBackendToString(config.transpose_comm_backend));
    printf("running using %s halo backend...\n", cudecompHaloCommBackendToString(config.halo_comm_backend));
  }

  // Allocating pencil memory

  // Get X-pencil information (with halo elements)
  cudecompPencilInfo_t pinfo_x;
  int32_t halo_extents_x[3]{1, 1, 1};
  CHECK_CUDECOMP_EXIT(cudecompGetPencilInfo(handle, grid_desc, &pinfo_x, 0, halo_extents_x, nullptr));

  // Get Y-pencil information
  cudecompPencilInfo_t pinfo_y;
  CHECK_CUDECOMP_EXIT(cudecompGetPencilInfo(handle, grid_desc, &pinfo_y, 1, nullptr, nullptr));

  // Get Z-pencil information
  cudecompPencilInfo_t pinfo_z;
  CHECK_CUDECOMP_EXIT(cudecompGetPencilInfo(handle, grid_desc, &pinfo_z, 2, nullptr, nullptr));

  // Allocate pencil memory
  int64_t data_num_elements = std::max(std::max(pinfo_x.size, pinfo_y.size), pinfo_z.size);

  // Allocate device buffer
  double* data_d;
  CHECK_CUDA_EXIT(hipMalloc(&data_d, data_num_elements * sizeof(*data_d)));

  // Allocate host buffer
  double* data = reinterpret_cast<double*>(malloc(data_num_elements * sizeof(*data)));

  // Initializing pencil data (host versions)

  // Access using a flattened loop
  for (int64_t l = 0; l < pinfo_x.size; ++l) {
    // Compute pencil-local coordinates, which are possibly in a permuted order.
    int i = l % pinfo_x.shape[0];
    int j = l / pinfo_x.shape[0] % pinfo_x.shape[1];
    int k = l / (pinfo_x.shape[0] * pinfo_x.shape[1]);

    // Compute global grid coordinates. To compute these, we offset the local coordinates
    // using the lower bound, lo, and use the order array to map the local coordinate order
    // to the global coordinate order.
    int gx[3];
    gx[pinfo_x.order[0]] = i + pinfo_x.lo[0];
    gx[pinfo_x.order[1]] = j + pinfo_x.lo[1];
    gx[pinfo_x.order[2]] = k + pinfo_x.lo[2];

    // Since the X-pencil also has halo elements, we apply an additional offset for the halo
    // elements in each direction, again using the order array to apply the extent to the
    // appropriate global coordinate.
    gx[pinfo_x.order[0]] -= pinfo_x.halo_extents[pinfo_x.order[0]];
    gx[pinfo_x.order[1]] -= pinfo_x.halo_extents[pinfo_x.order[1]];
    gx[pinfo_x.order[2]] -= pinfo_x.halo_extents[pinfo_x.order[2]];

    // Finally, we can set the buffer element, for example using a function based on the
    // global coordinates.
    data[l] = gx[0] + gx[1] + gx[2];
  }

  // Access using a triple loop with lo and hi
  int64_t l = 0;
  for (int k = pinfo_x.lo[2] - pinfo_x.halo_extents[pinfo_x.order[2]];
       k < pinfo_x.hi[2] + pinfo_x.halo_extents[pinfo_x.order[2]]; ++k) {
    for (int j = pinfo_x.lo[1] - pinfo_x.halo_extents[pinfo_x.order[1]];
         j < pinfo_x.hi[1] + pinfo_x.halo_extents[pinfo_x.order[1]]; ++j) {
      for (int i = pinfo_x.lo[0] - pinfo_x.halo_extents[pinfo_x.order[0]];
           i < pinfo_x.hi[0] + pinfo_x.halo_extents[pinfo_x.order[0]]; ++i) {

        // i, j, k are global coordinate values. Use order array to map to global
        // coordinate order.
        int gx[3];
        gx[pinfo_x.order[0]] = i;
        gx[pinfo_x.order[1]] = j;
        gx[pinfo_x.order[2]] = k;

        // Set the buffer element.
        data[l] = gx[0] + gx[1] + gx[2];
        l++;
      }
    }
  }

  // Copy host data to device
  CHECK_CUDA_EXIT(hipMemcpy(data_d, data, pinfo_x.size * sizeof(*data), hipMemcpyHostToDevice));

  // Initializing pencil data (device version using CUDA kernel)
  int threads_per_block = 256;
  int nblocks = (pinfo_x.size + threads_per_block - 1) / threads_per_block;
  initialize_pencil<<<nblocks, threads_per_block>>>(data_d, pinfo_x);

  // Allocating cuDecomp workspace

  // Get workspace sizes
  int64_t transpose_work_num_elements;
  CHECK_CUDECOMP_EXIT(cudecompGetTransposeWorkspaceSize(handle, grid_desc, &transpose_work_num_elements));

  int64_t halo_work_num_elements;
  CHECK_CUDECOMP_EXIT(
      cudecompGetHaloWorkspaceSize(handle, grid_desc, 0, pinfo_x.halo_extents, &halo_work_num_elements));

  // Allocate using cudecompMalloc
  int64_t dtype_size;
  CHECK_CUDECOMP_EXIT(cudecompGetDataTypeSize(CUDECOMP_DOUBLE, &dtype_size));

  double* transpose_work_d;
  CHECK_CUDECOMP_EXIT(cudecompMalloc(handle, grid_desc, reinterpret_cast<void**>(&transpose_work_d),
                                     transpose_work_num_elements * dtype_size));

  double* halo_work_d;
  CHECK_CUDECOMP_EXIT(
      cudecompMalloc(handle, grid_desc, reinterpret_cast<void**>(&halo_work_d), halo_work_num_elements * dtype_size));

  // Transposing data

  // Transpose from X-pencils to Y-pencils.
  CHECK_CUDECOMP_EXIT(cudecompTransposeXToY(handle, grid_desc, data_d, data_d, transpose_work_d, CUDECOMP_DOUBLE,
                                            pinfo_x.halo_extents, nullptr, nullptr, nullptr, 0));

  // Transpose from Y-pencils to Z-pencils.
  CHECK_CUDECOMP_EXIT(cudecompTransposeYToZ(handle, grid_desc, data_d, data_d, transpose_work_d, CUDECOMP_DOUBLE,
                                            nullptr, nullptr, nullptr, nullptr, 0));

  // Transpose from Z-pencils to Y-pencils.
  CHECK_CUDECOMP_EXIT(cudecompTransposeZToY(handle, grid_desc, data_d, data_d, transpose_work_d, CUDECOMP_DOUBLE,
                                            nullptr, nullptr, nullptr, nullptr, 0));

  // Transpose from Y-pencils to X-pencils.
  CHECK_CUDECOMP_EXIT(cudecompTransposeYToX(handle, grid_desc, data_d, data_d, transpose_work_d, CUDECOMP_DOUBLE,
                                            nullptr, pinfo_x.halo_extents, nullptr, nullptr, 0));

  // Updating halos

  // Setting for periodic halos in all directions
  bool halo_periods[3]{true, true, true};

  // Update X-pencil halos in X direction
  CHECK_CUDECOMP_EXIT(cudecompUpdateHalosX(handle, grid_desc, data_d, halo_work_d, CUDECOMP_DOUBLE,
                                           pinfo_x.halo_extents, halo_periods, 0, nullptr, 0));

  // Update X-pencil halos in Y direction
  CHECK_CUDECOMP_EXIT(cudecompUpdateHalosX(handle, grid_desc, data_d, halo_work_d, CUDECOMP_DOUBLE,
                                           pinfo_x.halo_extents, halo_periods, 1, nullptr, 0));

  // Update X-pencil halos in Z direction
  CHECK_CUDECOMP_EXIT(cudecompUpdateHalosX(handle, grid_desc, data_d, halo_work_d, CUDECOMP_DOUBLE,
                                           pinfo_x.halo_extents, halo_periods, 2, nullptr, 0));

  // Cleanup resources
  free(data);
  CHECK_CUDA_EXIT(hipFree(data_d));
  CHECK_CUDECOMP_EXIT(cudecompFree(handle, grid_desc, transpose_work_d));
  CHECK_CUDECOMP_EXIT(cudecompFree(handle, grid_desc, halo_work_d));
  CHECK_CUDECOMP_EXIT(cudecompGridDescDestroy(handle, grid_desc));
  CHECK_CUDECOMP_EXIT(cudecompFinalize(handle));

  CHECK_MPI_EXIT(MPI_Finalize());
}
